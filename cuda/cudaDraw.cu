#include "hip/hip_runtime.h"
#include "cudaDraw.h"
#include "cudaAlphaBlend.cuh"


#define MIN(a,b)  (a < b ? a : b)
#define MAX(a,b)  (a > b ? a : b)

template<typename T> inline __device__ __host__ T sqr(T x) 				    { return x*x; }

inline __device__ __host__ float dist2(float x1, float y1, float x2, float y2) { return sqr(x1-x2) + sqr(y1-y2); }
inline __device__ __host__ float dist(float x1, float y1, float x2, float y2)  { return sqrtf(dist2(x1,y1,x2,y2)); }

//----------------------------------------------------------------------------
// Line drawing (find if the distance to the line <= line_width)
// Distance from point to line segment - https://stackoverflow.com/a/1501725
//----------------------------------------------------------------------------
inline __device__ float lineDistanceSquared(float x, float y, float x1, float y1, float x2, float y2) 
{
	const float d = dist2(x1, y1, x2, y2);
	const float t = ((x-x1) * (x2-x1) + (y-y1) * (y2-y1)) / d;
	const float u = MAX(0, MIN(1, t));
	
	return dist2(x, y, x1 + u * (x2 - x1), y1 + u * (y2 - y1));
}



template<typename T>
__global__ void gpuDrawLine( T* img, int imgWidth, int imgHeight, int offset_x, int offset_y, int x1, int y1, int x2, int y2, const float4 color, float line_width2 ) 
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x + offset_x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y + offset_y;

	if( x >= imgWidth || y >= imgHeight || x < 0 || y < 0 )
		return;

	if( lineDistanceSquared(x, y, x1, y1, x2, y2) <= line_width2 )
	{
		const int idx = y * imgWidth + x;
		img[idx] = cudaAlphaBlend(img[idx], color);
	}
}




template<typename T>
__global__ void gpuDrawLineYUYV( T* img, int imgWidth, int imgHeight, int offset_x, int offset_y, 
int x1, int y1, int x2, int y2, uint8_t color_y,uint8_t color_u,uint8_t color_v, float line_width2 ) 
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x + offset_x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y + offset_y;

	if( x >= imgWidth || y >= imgHeight || x < 0 || y < 0 )
		return;

	if( lineDistanceSquared(x, y, x1, y1, x2, y2) <= line_width2 )
	{
		// const int idx = y * imgWidth + x;
		
		// img[idx] = cudaAlphaBlend(img[idx], color);


		// img[y*2*imgWidth+4*(x/2)+0] = color_u;
		// img[y*2*imgWidth+4*(x/2)+1] = color_y;
		// img[y*2*imgWidth+4*(x/2)+2] = color_v;
		// img[y*2*imgWidth+4*(x/2)+3] = color_y;


		img[y*2*imgWidth+2*x] = color_y;
		img[y*2*imgWidth+4*(x/2)+1] = color_u;
		img[y*2*imgWidth+4*(x/2)+3] = color_v;
	}
}

// cudaDrawLine
hipError_t cudaDrawLine( void* input, void* output, size_t width, size_t height, imageFormat format, int x1, int y1, int x2, int y2, const float4& color, float line_width )
{
	if( !input || !output || width == 0 || height == 0 || line_width <= 0 )
		return hipErrorInvalidValue;
	
	// check for lines < 2 pixels in length
	if( dist(x1,y1,x2,y2) < 2.0 )
	{
		LogWarning(LOG_CUDA "cudaDrawLine() - line has length < 2, skipping (%i,%i) (%i,%i)\n", x1, y1, x2, y2);
		return hipSuccess;
	}
	
	// if the input and output images are different, copy the input to the output
	// this is because we only launch the kernel in the approximate area of the circle
	if( input != output )
		CUDA(hipMemcpy(output, input, imageFormatSize(format, width, height), hipMemcpyDeviceToDevice));
		
	// find a box around the line
	const int left = MIN(x1,x2) - line_width;
	const int right = MAX(x1,x2) + line_width;
	const int top = MIN(y1,y2) - line_width;
	const int bottom = MAX(y1,y2) + line_width;

	// launch kernel
	const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(right - left, blockDim.x), iDivUp(bottom - top, blockDim.y));

	#define LAUNCH_DRAW_LINE(type) \
		gpuDrawLine<type><<<gridDim, blockDim>>>((type*)output, width, height, left, top, x1, y1, x2, y2, color, line_width * line_width)
	
	if( format == IMAGE_RGB8 )
		LAUNCH_DRAW_LINE(uchar3);
	else if( format == IMAGE_RGBA8 )
		LAUNCH_DRAW_LINE(uchar4);
	else if( format == IMAGE_RGB32F )
		LAUNCH_DRAW_LINE(float3); 
	else if( format == IMAGE_RGBA32F )
		LAUNCH_DRAW_LINE(float4);
	else if( format == IMAGE_YUYV ) {
		uint8_t color_y = static_cast<uint8_t>(((int)(30 * color.x) + (int)(59 * color.y) + (int)(11 * color.z)) / 100);
		uint8_t color_u = static_cast<uint8_t>(((int)(-17 * color.x) - (int)(33 * color.y) + (int)(50 * color.z) + 12800) / 100);
		uint8_t color_v = static_cast<uint8_t>(((int)(50 * color.x) - (int)(42 * color.y) - (int)(8 * color.z) + 12800) / 100);
		gpuDrawLineYUYV<unsigned char><<<gridDim, blockDim>>>((unsigned char *)output,width,height,left,top,x1,y1,x2,y2,
															color_y,color_u,color_v, line_width*line_width) ;
	}
	else {
		imageFormatErrorMsg(LOG_CUDA, "cudaDrawLine()", format);
		return hipErrorInvalidValue;
	}
		
	return hipGetLastError();
}





__global__ void gpuDeinterlace( unsigned char* input_cur, unsigned char* output, size_t width, size_t height)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	int pitch = 1536; //1536;// this is very important and not necesserly is equal to 2*width !!

	// if(input_cur[720*2+2] == 0)
	// 	output[2*y*pitch + x] =  static_cast<unsigned char>( ( (int) input_cur[y*pitch+x] + (int) output[(2*y+1)*pitch + x] )/2 ); 

	// else if(input_cur[720*2+2] == 255)
	// 	output[(2*y+1)*pitch + x] =  static_cast<unsigned char>( ( (int) input_cur[y*pitch+x] + (int) output[2*y*pitch + x] )/2) ;

	// if(input_cur[720*2+2] == 0)
	// 	output[2*y*pitch + x] =  static_cast<unsigned char>(  input_cur[y*pitch+x]); 

	// else if(input_cur[720*2+2] == 255)
	// 	output[(2*y+1)*pitch + x] =  static_cast<unsigned char>( input_cur[y*pitch+x] ) ;
	
	// if( x == 0 && y ==0 ) {
	// 	printf("field 0:%d \t",	input_cur[738*2+0] );
	// 	printf(" | 1:%d \t",	input_cur[721*2+1] );
	// 	printf(" | 2:%d \t",	input_cur[721*2+2] );
	// 	printf(" | 3:%d \n",	input_cur[721*2+3] );
	// }

	
	
	unsigned char field_flag[1];//

	field_flag[0] = input_cur[721*2+2];

	if(field_flag[0] == 0){
		output[2*y*pitch + x] = static_cast<unsigned char>(input_cur[y*pitch+x] ); 
		// output[(2*y+1)*pitch + x] = static_cast<unsigned char>( ( (int) input_cur[y*pitch+x] + (int) output[(2*y+1)*pitch + x] )/2) ;
	}
	else if(field_flag[0] == 255) {
		output[(2*y+1)*pitch + x] =  static_cast<unsigned char>(input_cur[y*pitch+x] );
		// output[(2*y)*pitch + x] = static_cast<unsigned char>( ( (int) input_cur[y*pitch+x] + (int) output[(2*y)*pitch + x] )/2) ;
	}

}

hipError_t cudaDeinterlace( void* input_cur, void* output, size_t width, size_t height)
{	
	const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(2*720,blockDim.x), iDivUp(288,blockDim.y)); // 736 * 288
	gpuDeinterlace<<<gridDim, blockDim>>>((unsigned char *) input_cur, 
										(unsigned char *) output,width,height);// TODO: add pitch

	return hipGetLastError();
}
